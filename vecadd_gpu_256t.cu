
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
 
// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
 int index = threadIdx.x;
 int stride = blockDim.x;
 for(int i = index; i < n; i += stride)
	y[i] = x[i]+y[i];
}
 
int main(void)
{
 int N = 1<<29; // 1M elements
 
// Allocating unified memory
float *x, *y;
hipMallocManaged(&x, N*sizeof(float));
hipMallocManaged(&y, N*sizeof(float));

 // initialize x and y arrays on the st
 for (int i = 0; i < N; i++) {
   x[i] = 1.0f;
   y[i] = 2.0f;
 }

add<<<1,256>>>(N, x, y);

hipDeviceSynchronize();

 // Check for errors (all values should be 3.0f)
 float maxError = 0.0f;
 for (int i = 0; i < N; i++)
   maxError = fmax(maxError, fabs(y[i]-3.0f));
 std::cout << "Max error: " << maxError << std::endl;
 
 // Free memory
 hipFree(x);
 hipFree(y);
 
 return 0;
}
