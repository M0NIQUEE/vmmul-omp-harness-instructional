
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
 
// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
 int index = blockIdx.x * blockDim.x + threadIdx.x;
 int stride = blockDim.x * gridDim.x;
 for(int i = index; i < n; i += stride)
	y[i] = x[i]+y[i];
}
 
int main(void)
{
 int N = 1<<29; // 1M elements
 
// Allocating unified memory
float *x, *y;
hipMallocManaged(&x, N*sizeof(float));
hipMallocManaged(&y, N*sizeof(float));

 // initialize x and y arrays on the st
 for (int i = 0; i < N; i++) {
   x[i] = 1.0f;
   y[i] = 2.0f;
 }

int blockSize = 256;
int numBlocks = (N + blockSize -1)/ blockSize;

std::cout<<"Number of thread blocks used in this run: " << blockSize;

add<<<numBlocks, blockSize>>>(N, x, y);


hipDeviceSynchronize();

 // Check for errors (all values should be 3.0f)
 float maxError = 0.0f;
 for (int i = 0; i < N; i++)
   maxError = fmax(maxError, fabs(y[i]-3.0f));
 std::cout << "Max error: " << maxError << std::endl;
 
 // Free memory
 hipFree(x);
 hipFree(y);
 
 return 0;
}
